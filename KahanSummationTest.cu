#include "hip/hip_runtime.h"

#include "KahanSumSequence.cuh"

#include <hip/hip_runtime.h>

template <typename T>
struct GeometricProgression {
	__device__ T operator()(size_t counter, T _) {
		T tmp = current; current *= q; return tmp;
	}

	__device__ GeometricProgression(T q, T q0): q(q), q0(q0), current(q0) { }

	__device__ void reset() { current = q0; }
private:
	T const q;
	T const q0;
	T current;
};

template <typename T>
__global__ void test_kernell(T *res) {
	*res = iki::math::device::kahan_summation_sequence<T, GeometricProgression<T>>(GeometricProgression<T>(T(0.5),T(0.5)), 10);
}

#include <iostream>

float run_kernell() {
	using namespace std;

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		cout << "Error in starting cuda device" << endl;
		goto Error;
	}
	{
		float *res_dev_ptr; float res_host = -1.;
		cudaStatus = hipMalloc((void **)&res_dev_ptr, sizeof(float));

		test_kernell<float> <<<1, 1>>> (res_dev_ptr);
		hipMemcpy(&res_host, res_dev_ptr, sizeof(float), hipMemcpyDeviceToHost);
		cout << "Res: " << res_host << endl;

		hipFree(res_dev_ptr);
	}

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		cout << "Error in starting cuda device" << endl;
		goto Error;
	}

Error:
	return -1.f;
}